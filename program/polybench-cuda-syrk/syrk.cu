#include "hip/hip_runtime.h"
/**
 * syrk.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 *
 * Updated by Grigori Fursin (http://cTuning.org/lab/people/gfursin)
 * to work with Collective Mind, OpenME plugin interface and 
 * Collective Knowledge Frameworks for automatic, machine-learning based
 * and collective tuning and data mining: http://cTuning.org
 *
 */

#ifndef WINDOWS
 #include <unistd.h>
#endif

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include "polybench.h"

#ifdef OPENME
#include <openme.h>
#endif
#ifdef XOPENME
#include <xopenme.h>
#endif

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size */
#ifndef NI
#define NI 256 // 1024
#endif
#ifndef NJ
#define NJ 256 // 1024
#endif

/* Thread block dimensions */
#ifndef DIM_THREAD_BLOCK_Y
#define DIM_THREAD_BLOCK_X 32
#endif
#ifndef DIM_THREAD_BLOCK_Y
#define DIM_THREAD_BLOCK_Y 8
#endif

/* Declared constant values for alpha and beta (same as values in PolyBench 2.0) */
#define alpha 12435
#define beta 4546

/* Can switch DATA_TYPE between float and double */
# ifndef DATA_TYPE
#  define DATA_TYPE float
# endif

void init_arrays(DATA_TYPE* A, DATA_TYPE* C)
{
	int i, j;
	
	for (i = 0; i < NJ; i++)
    	{
		for (j = 0; j < NI; j++)
		{
			A[i*NI + j] = ((DATA_TYPE) i*j) / NJ;
		}
		
		for (j = 0; j < NJ; j++)
		{
			C[i*NI + j] = ((DATA_TYPE) i*j + 2) / NJ;
		}
	}
}


void syrk(DATA_TYPE* A, DATA_TYPE* C)
{
	int i, j, k;
	
	/*  C := alpha*A*A' + beta*C */
	for (i = 0; i < NJ; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			C[i*NI + j] *= beta;
		}
	}
	
	for (i = 0; i < NJ; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			for (k = 0; k < NI; k++)
			{
				C[i*NJ + j] += alpha * A[i*NI + k] * A[j*NI + k];
			}
		}
	}
}


void compareResults(DATA_TYPE* C, DATA_TYPE* C_outputFromGpu)
{
	int i,j,fail;
	fail = 0;

	// Compare C with D
	for (i=0; i<NJ; i++)
	{
		for (j=0; j<NI; j++)
		{
			if (percentDiff(C[i*NI + j], C_outputFromGpu[i*NI + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{
				fail++;
			}
		}
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
  /* Grigori Fursin added support for CK widgets */
  int gpgpu_device_id=GPU_DEVICE;

  int devID = 0;
  hipError_t error;
  hipDeviceProp_t deviceProp;
  error = hipGetDevice(&devID);

  if (getenv("CK_COMPUTE_DEVICE_ID")!=NULL) gpgpu_device_id=atol(getenv("CK_COMPUTE_DEVICE_ID"));

  hipGetDeviceProperties(&deviceProp, gpgpu_device_id);

  if (deviceProp.computeMode == hipComputeModeProhibited)
  {
    printf("Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
    exit(EXIT_SUCCESS);
  }

  if (error != hipSuccess)
    printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
  else
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);

  hipSetDevice( gpgpu_device_id );
}


__global__ void syrk_kernel(DATA_TYPE ALPHA, DATA_TYPE BETA, DATA_TYPE *a, DATA_TYPE *c)
{
	/*  C := alpha*A*A' + beta*C */
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NJ) && (j < NJ))
	{
		c[i * NJ + j] *= beta;
		int k;		
		for(k=0; k< NI; k++)
		{
			c[i * NJ + j] += alpha * a[i * NI + k] * a[j * NI + k];
		}
	}
}


void syrkCuda(DATA_TYPE* A, DATA_TYPE* C, DATA_TYPE* C_outputFromGpu)
{
        hipError_t error;
	double t_start, t_end;

	DATA_TYPE* A_gpu;
	DATA_TYPE* C_gpu;

	error=hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NJ * NI);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NJ * NJ);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NJ * NI, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NJ * NJ, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)(ceil(((float)NJ) / ((float)DIM_THREAD_BLOCK_X))), (size_t)ceil(((float)NJ) / ((float)DIM_THREAD_BLOCK_Y)));
//	t_start = rtclock();
	syrk_kernel<<<grid,block>>>(alpha, beta, A_gpu,C_gpu);
	hipDeviceSynchronize();
//	t_end = rtclock();
//	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

	error=hipMemcpy(C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * NJ * NJ, hipMemcpyDeviceToHost);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	hipFree(A_gpu);
	hipFree(C_gpu);
}


int main()
{
  /* Prepare ctuning vars */
  long ct_repeat=0;
  long ct_repeat_max=1;

  double t_start, t_end;

  DATA_TYPE* A;
  DATA_TYPE* C;
  DATA_TYPE* C_outputFromGpu;

#ifdef XOPENME
  xopenme_init(2,0);
#endif

#ifdef OPENME
  openme_init(NULL,NULL,NULL,0);
  openme_callback("PROGRAM_START", NULL);
#endif

  /* Run kernel. */
  if (getenv("CT_REPEAT_MAIN")!=NULL) ct_repeat_max=atol(getenv("CT_REPEAT_MAIN"));

  A = (DATA_TYPE*)malloc(NJ*NI*sizeof(DATA_TYPE));
  C = (DATA_TYPE*)malloc(NJ*NI*sizeof(DATA_TYPE));
  C_outputFromGpu = (DATA_TYPE*)malloc(NJ*NI*sizeof(DATA_TYPE));

  srand(1);
  init_arrays(A, C);
  GPU_argv_init();	

#ifdef OPENME
  openme_callback("ACC_KERNEL_START", NULL);
#endif
#ifdef XOPENME
  xopenme_clock_start(0);
#endif
  for (ct_repeat=0; ct_repeat<ct_repeat_max; ct_repeat++)
  {
    syrkCuda(A, C, C_outputFromGpu);
  }
#ifdef XOPENME
  xopenme_clock_end(0);
#endif
#ifdef OPENME
  openme_callback("ACC_KERNEL_END", NULL);
#endif

/*
  srand(1);
  init_arrays(A, C);

#ifdef OPENME
  openme_callback("KERNEL_START", NULL);
#endif
#ifdef XOPENME
  xopenme_clock_start(1);
#endif
  for (ct_repeat=0; ct_repeat<ct_repeat_max; ct_repeat++)
  {
    syrk(A, C);
  }
#ifdef XOPENME
  xopenme_clock_end(1);
#endif
#ifdef OPENME
  openme_callback("KERNEL_END", NULL);
#endif
*/

  compareResults(C, C_outputFromGpu);

  free(A);
  free(C);
  free(C_outputFromGpu);

#ifdef XOPENME
  xopenme_dump_state();
  xopenme_finish();
#endif

#ifdef OPENME
  openme_callback("PROGRAM_END", NULL);
#endif

  return 0;
}

