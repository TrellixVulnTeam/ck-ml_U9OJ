#include "hip/hip_runtime.h"
/**
 * mvt.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 *
 * Updated by Grigori Fursin (http://cTuning.org/lab/people/gfursin)
 * to work with Collective Mind, OpenME plugin interface and 
 * Collective Knowledge Frameworks for automatic, machine-learning based
 * and collective tuning and data mining: http://cTuning.org
 *
 */

#ifndef WINDOWS
 #include <unistd.h>
#endif

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include "polybench.h"

#ifdef OPENME
#include <openme.h>
#endif
#ifdef XOPENME
#include <xopenme.h>
#endif

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size */
#ifndef NJ
#define NJ 1024
#endif

/* Thread block dimensions */
#ifndef DIM_THREAD_BLOCK_X
#define DIM_THREAD_BLOCK_X 256
#endif
#ifndef DIM_THREAD_BLOCK_Y
#define DIM_THREAD_BLOCK_Y 1
#endif

/* Can switch DATA_TYPE between float and double */
# ifndef DATA_TYPE
#  define DATA_TYPE float
# endif

void init_array(DATA_TYPE* A, DATA_TYPE* x1, DATA_TYPE* x2, DATA_TYPE* y1, DATA_TYPE* y2)
{
	int i, j;

	for (i = 0; i < NJ; i++)
	{
		x1[i] = ((DATA_TYPE) i) / NJ;
		x2[i] = ((DATA_TYPE) i + 1) / NJ;
		y1[i] = ((DATA_TYPE) i + 3) / NJ;
		y2[i] = ((DATA_TYPE) i + 4) / NJ;
		for (j = 0; j < NJ; j++)
		{
			A[i*NJ + j] = ((DATA_TYPE) i*j) / NJ;
		}
	}
}



void runMvt(DATA_TYPE* a, DATA_TYPE* x1, DATA_TYPE* x2, DATA_TYPE* y1, DATA_TYPE* y2)
{
	int i, j;
	
	for (i=0; i<NJ; i++) 
	{
		x1[i]=0;
		for (j=0; j<NJ; j++) 
		{
       			x1[i] = x1[i] + a[i*NJ + j] * y1[j];
        	}
    	}
	
	for (i=0; i<NJ; i++) 
	{
		x2[i]=0;
		for (j=0; j<NJ; j++) 
		{
 		       	x2[i] = x2[i] + a[j*NJ + i] * y2[j];
      		}
    	}
}


void compareResults(DATA_TYPE* x1, DATA_TYPE* x1_outputFromGpu, DATA_TYPE* x2, DATA_TYPE* x2_outputFromGpu)
{
	int i, fail;
	fail = 0;
	
	for (i=0; i<NJ; i++) 
	{
		if (percentDiff(x1[i], x1_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
		{
			fail++;
		}

		if (percentDiff(x2[i], x2_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
		{
			fail++;
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
  /* Grigori Fursin added support for CK widgets */
  int gpgpu_device_id=GPU_DEVICE;

  int devID = 0;
  hipError_t error;
  hipDeviceProp_t deviceProp;
  error = hipGetDevice(&devID);

  if (getenv("CK_COMPUTE_DEVICE_ID")!=NULL) gpgpu_device_id=atol(getenv("CK_COMPUTE_DEVICE_ID"));

  hipGetDeviceProperties(&deviceProp, gpgpu_device_id);

  if (deviceProp.computeMode == hipComputeModeProhibited)
  {
    printf("Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
    exit(EXIT_SUCCESS);
  }

  if (error != hipSuccess)
    printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
  else
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);

  hipSetDevice( gpgpu_device_id );
}


__global__ void mvt_kernel1(DATA_TYPE *a, DATA_TYPE *x1, DATA_TYPE *y_1)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < NJ)
	{
		int j;
		for(j=0; j < NJ; j++)
		{
			x1[i] += a[i * NJ + j] * y_1[j];
		}
	}
}


__global__ void mvt_kernel2(DATA_TYPE *a, DATA_TYPE *x2, DATA_TYPE *y_2)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < NJ)
	{
		int j;
		for(j=0; j < NJ; j++)
		{
			x2[i] += a[j * NJ + i] * y_2[j];	
		}
	}
}

void mvtCuda(DATA_TYPE* a, DATA_TYPE* x1, DATA_TYPE* x2, DATA_TYPE* y_1, DATA_TYPE* y_2, 
			DATA_TYPE* x1_outputFromGpu, DATA_TYPE* x2_outputFromGpu)
{
        hipError_t error;
	double t_start, t_end;

	DATA_TYPE* a_gpu;
	DATA_TYPE* x1_gpu;
	DATA_TYPE* x2_gpu;
	DATA_TYPE* y_1_gpu;
	DATA_TYPE* y_2_gpu;

	error=hipMalloc((void **)&a_gpu, sizeof(DATA_TYPE) * NJ * NJ);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&x1_gpu, sizeof(DATA_TYPE) * NJ);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&x2_gpu, sizeof(DATA_TYPE) * NJ);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&y_1_gpu, sizeof(DATA_TYPE) * NJ);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&y_2_gpu, sizeof(DATA_TYPE) * NJ);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(a_gpu, a, sizeof(DATA_TYPE) * NJ * NJ, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(x1_gpu, x1, sizeof(DATA_TYPE) * NJ, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(x2_gpu, x2, sizeof(DATA_TYPE) * NJ, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(y_1_gpu, y_1, sizeof(DATA_TYPE) * NJ, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(y_2_gpu, y_2, sizeof(DATA_TYPE) * NJ, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)ceil((float)NJ/ ((float)DIM_THREAD_BLOCK_X)), 1);
	
//	t_start = rtclock();
	mvt_kernel1<<<grid,block>>>(a_gpu,x1_gpu,y_1_gpu);
	mvt_kernel2<<<grid,block>>>(a_gpu,x2_gpu,y_2_gpu);
	hipDeviceSynchronize();
//	t_end = rtclock();
//	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

	error=hipMemcpy(x1_outputFromGpu, x1_gpu, sizeof(DATA_TYPE) * NJ, hipMemcpyDeviceToHost);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(x2_outputFromGpu, x2_gpu, sizeof(DATA_TYPE) * NJ, hipMemcpyDeviceToHost);    
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }
	
	hipFree(a_gpu);
	hipFree(x1_gpu);
	hipFree(x2_gpu);
	hipFree(y_1_gpu);
	hipFree(y_2_gpu);
}


int main()
{
  /* Prepare ctuning vars */
  long ct_repeat=0;
  long ct_repeat_max=1;

  DATA_TYPE* a;
  DATA_TYPE* x1;
  DATA_TYPE* x2;
  DATA_TYPE* x1_outputFromGpu;
  DATA_TYPE* x2_outputFromGpu;
  DATA_TYPE* y_1;
  DATA_TYPE* y_2;

#ifdef XOPENME
  xopenme_init(2,0);
#endif

#ifdef OPENME
  openme_init(NULL,NULL,NULL,0);
  openme_callback("PROGRAM_START", NULL);
#endif

  /* Run kernel. */
  if (getenv("CT_REPEAT_MAIN")!=NULL) ct_repeat_max=atol(getenv("CT_REPEAT_MAIN"));

  a = (DATA_TYPE*)malloc(NJ*NJ*sizeof(DATA_TYPE));
  x1 = (DATA_TYPE*)malloc(NJ*sizeof(DATA_TYPE));
  x2 = (DATA_TYPE*)malloc(NJ*sizeof(DATA_TYPE));
  x1_outputFromGpu = (DATA_TYPE*)malloc(NJ*sizeof(DATA_TYPE));
  x2_outputFromGpu = (DATA_TYPE*)malloc(NJ*sizeof(DATA_TYPE));
  y_1 = (DATA_TYPE*)malloc(NJ*sizeof(DATA_TYPE));
  y_2 = (DATA_TYPE*)malloc(NJ*sizeof(DATA_TYPE));

  srand(1);
  init_array(a, x1, x2, y_1, y_2);
  GPU_argv_init();

#ifdef OPENME
  openme_callback("ACC_KERNEL_START", NULL);
#endif
#ifdef XOPENME
  xopenme_clock_start(0);
#endif
  for (ct_repeat=0; ct_repeat<ct_repeat_max; ct_repeat++)
  {
    mvtCuda(a, x1, x2, y_1, y_2, x1_outputFromGpu, x2_outputFromGpu);
  }
#ifdef XOPENME
  xopenme_clock_end(0);
#endif
#ifdef OPENME
  openme_callback("ACC_KERNEL_END", NULL);
#endif

/*
   srand(1);
  init_array(a, x1, x2, y_1, y_2);

#ifdef OPENME
  openme_callback("KERNEL_START", NULL);
#endif
#ifdef XOPENME
  xopenme_clock_start(1);
#endif
  for (ct_repeat=0; ct_repeat<ct_repeat_max; ct_repeat++)
  {
    runMvt(a, x1, x2, y_1, y_2);
  }
#ifdef XOPENME
  xopenme_clock_end(1);
#endif
#ifdef OPENME
  openme_callback("KERNEL_END", NULL);
#endif
*/

  compareResults(x1, x1_outputFromGpu, x2, x2_outputFromGpu);

  free(a);
  free(x1);
  free(x2);
  free(x1_outputFromGpu);
  free(x2_outputFromGpu);
  free(y_1);
  free(y_2);

#ifdef XOPENME
  xopenme_dump_state();
  xopenme_finish();
#endif

#ifdef OPENME
  openme_callback("PROGRAM_END", NULL);
#endif

  return 0;
}

