#include "hip/hip_runtime.h"
/**
 * fdtd2d.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 *
 * Updated by Grigori Fursin (http://cTuning.org/lab/people/gfursin)
 * to work with Collective Mind, OpenME plugin interface and 
 * Collective Knowledge Frameworks for automatic, machine-learning based
 * and collective tuning and data mining: http://cTuning.org
 *
 */

#ifndef WINDOWS
 #include <unistd.h>
#endif

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include "polybench.h"

#ifdef OPENME
#include <openme.h>
#endif
#ifdef XOPENME
#include <xopenme.h>
#endif

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 10.05

#define GPU_DEVICE 0

/* Problem size */
#ifndef tmax
#define tmax 500
#endif
#ifndef NX
#define NX 512 // 2048
#endif
#ifndef NY
#define NY 512 // 2048
#endif

/* Thread block dimensions */
#ifndef DIM_THREAD_BLOCK_X
#define DIM_THREAD_BLOCK_X 32
#endif
#ifndef DIM_THREAD_BLOCK_Y
#define DIM_THREAD_BLOCK_Y 8
#endif

/* Can switch DATA_TYPE between float and double */
# ifndef DATA_TYPE
#  define DATA_TYPE float
# endif

void init_arrays(DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz)
{
	int i, j;

  	for (i = 0; i < tmax; i++)
	{
		_fict_[i] = (DATA_TYPE) i;
	}
	
	for (i = 0; i < NX; i++)
	{
		for (j = 0; j < NY; j++)
		{
			ex[i*NY + j] = ((DATA_TYPE) i*(j+1) + 1) / NX;
			ey[i*NY + j] = ((DATA_TYPE) (i-1)*(j+2) + 2) / NX;
			hz[i*NY + j] = ((DATA_TYPE) (i-9)*(j+4) + 3) / NX;
		}
	}
}


void runFdtd(DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz)
{
	int t, i, j;
	
	for (t=0; t < tmax; t++)  
	{
		for (j=0; j < NY; j++)
		{
			ey[0*NY + j] = _fict_[t];
		}
	
		for (i = 1; i < NX; i++)
		{
       		for (j = 0; j < NY; j++)
			{
       			ey[i*NY + j] = ey[i*NY + j] - 0.5*(hz[i*NY + j] - hz[(i-1)*NY + j]);
        		}
		}

		for (i = 0; i < NX; i++)
		{
       		for (j = 1; j < NY; j++)
			{
				ex[i*(NY+1) + j] = ex[i*(NY+1) + j] - 0.5*(hz[i*NY + j] - hz[i*NY + (j-1)]);
			}
		}

		for (i = 0; i < NX; i++)
		{
			for (j = 0; j < NY; j++)
			{
				hz[i*NY + j] = hz[i*NY + j] - 0.7*(ex[i*(NY+1) + (j+1)] - ex[i*(NY+1) + j] + ey[(i+1)*NY + j] - ey[i*NY + j]);
			}
		}
	}
}


void compareResults(DATA_TYPE* hz1, DATA_TYPE* hz2)
{
	int i, j, fail;
	fail = 0;
	
	for (i=0; i < NX; i++) 
	{
		for (j=0; j < NY; j++) 
		{
			if (percentDiff(hz1[i*NY + j], hz2[i*NY + j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
				fail++;
			}
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
  /* Grigori Fursin added support for CK widgets */
  int gpgpu_device_id=GPU_DEVICE;

  int devID = 0;
  hipError_t error;
  hipDeviceProp_t deviceProp;
  error = hipGetDevice(&devID);

  if (getenv("CK_COMPUTE_DEVICE_ID")!=NULL) gpgpu_device_id=atol(getenv("CK_COMPUTE_DEVICE_ID"));

  hipGetDeviceProperties(&deviceProp, gpgpu_device_id);

  if (deviceProp.computeMode == hipComputeModeProhibited)
  {
    printf("Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
    exit(EXIT_SUCCESS);
  }

  if (error != hipSuccess)
    printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
  else
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);

  hipSetDevice( gpgpu_device_id );
}



__global__ void fdtd_step1_kernel(DATA_TYPE* _fict_, DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NX) && (j < NY))
	{
		if (i == 0) 
		{
			ey[i * NY + j] = _fict_[t];
		}
		else
		{ 
			ey[i * NY + j] = ey[i * NY + j] - 0.5f*(hz[i * NY + j] - hz[(i-1) * NY + j]);
		}
	}
}



__global__ void fdtd_step2_kernel(DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i < NX) && (j < NY) && (j > 0))
	{
		ex[i * (NY+1) + j] = ex[i * (NY+1) + j] - 0.5f*(hz[i * NY + j] - hz[i * NY + (j-1)]);
	}
}


__global__ void fdtd_step3_kernel(DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i < NX) && (j < NY))
	{	
		hz[i * NY + j] = hz[i * NY + j] - 0.7f*(ex[i * (NY+1) + (j+1)] - ex[i * (NY+1) + j] + ey[(i + 1) * NY + j] - ey[i * NY + j]);
	}
}


void fdtdCuda(DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz, DATA_TYPE* hz_outputFromGpu)
{
        hipError_t error;
	double t_start, t_end;

	DATA_TYPE *_fict_gpu;
	DATA_TYPE *ex_gpu;
	DATA_TYPE *ey_gpu;
	DATA_TYPE *hz_gpu;

	error=hipMalloc((void **)&_fict_gpu, sizeof(DATA_TYPE) * tmax);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&ex_gpu, sizeof(DATA_TYPE) * NX * (NY + 1));
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&ey_gpu, sizeof(DATA_TYPE) * (NX + 1) * NY);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&hz_gpu, sizeof(DATA_TYPE) * NX * NY);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }


	error=hipMemcpy(_fict_gpu, _fict_, sizeof(DATA_TYPE) * tmax, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(ex_gpu, ex, sizeof(DATA_TYPE) * NX * (NY + 1), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(ey_gpu, ey, sizeof(DATA_TYPE) * (NX + 1) * NY, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(hz_gpu, hz, sizeof(DATA_TYPE) * NX * NY, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid( (size_t)ceil(((float)NY) / ((float)block.x)), (size_t)ceil(((float)NX) / ((float)block.y)));

//	t_start = rtclock();

	for(int t = 0; t< tmax; t++)
	{
		fdtd_step1_kernel<<<grid,block>>>(_fict_gpu, ex_gpu, ey_gpu, hz_gpu, t);
		hipDeviceSynchronize();
		fdtd_step2_kernel<<<grid,block>>>(ex_gpu, ey_gpu, hz_gpu, t);
		hipDeviceSynchronize();
		fdtd_step3_kernel<<<grid,block>>>(ex_gpu, ey_gpu, hz_gpu, t);
		hipDeviceSynchronize();
	}
	
//	t_end = rtclock();
//    	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

	error=hipMemcpy(hz_outputFromGpu, hz_gpu, sizeof(DATA_TYPE) * NX * NY, hipMemcpyDeviceToHost);	
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	hipFree(_fict_gpu);
	hipFree(ex_gpu);
	hipFree(ey_gpu);
	hipFree(hz_gpu);
}


int main()
{
  /* Prepare ctuning vars */
  long ct_repeat=0;
  long ct_repeat_max=1;

  double t_start, t_end;

  DATA_TYPE* _fict_;
  DATA_TYPE* ex;
  DATA_TYPE* ey;
  DATA_TYPE* hz;
  DATA_TYPE* hz_outputFromGpu;

#ifdef XOPENME
  xopenme_init(2,0);
#endif

#ifdef OPENME
  openme_init(NULL,NULL,NULL,0);
  openme_callback("PROGRAM_START", NULL);
#endif

  /* Run kernel. */
  if (getenv("CT_REPEAT_MAIN")!=NULL) ct_repeat_max=atol(getenv("CT_REPEAT_MAIN"));

  _fict_ = (DATA_TYPE*)malloc(tmax*sizeof(DATA_TYPE));
  ex = (DATA_TYPE*)malloc(NX*(NY+1)*sizeof(DATA_TYPE));
  ey = (DATA_TYPE*)malloc((NX+1)*NY*sizeof(DATA_TYPE));
  hz = (DATA_TYPE*)malloc(NX*NY*sizeof(DATA_TYPE));
  hz_outputFromGpu = (DATA_TYPE*)malloc(NX*NY*sizeof(DATA_TYPE));

  srand(1);
  init_arrays(_fict_, ex, ey, hz);
  GPU_argv_init();

#ifdef OPENME
  openme_callback("ACC_KERNEL_START", NULL);
#endif
#ifdef XOPENME
  xopenme_clock_start(0);
#endif
  for (ct_repeat=0; ct_repeat<ct_repeat_max; ct_repeat++)
  {
    fdtdCuda(_fict_, ex, ey, hz, hz_outputFromGpu);
  }
#ifdef XOPENME
  xopenme_clock_end(0);
#endif
#ifdef OPENME
  openme_callback("ACC_KERNEL_END", NULL);
#endif

/*
  srand(1);
  init_arrays(_fict_, ex, ey, hz);

#ifdef OPENME
  openme_callback("KERNEL_START", NULL);
#endif
#ifdef XOPENME
  xopenme_clock_start(1);
#endif
  for (ct_repeat=0; ct_repeat<ct_repeat_max; ct_repeat++)
  {
    runFdtd(_fict_, ex, ey, hz);
  }
#ifdef XOPENME
  xopenme_clock_end(1);
#endif
#ifdef OPENME
  openme_callback("KERNEL_END", NULL);
#endif
*/

  compareResults(hz, hz_outputFromGpu);

  free(_fict_);
  free(ex);
  free(ey);
  free(hz);
  free(hz_outputFromGpu);

#ifdef XOPENME
  xopenme_dump_state();
  xopenme_finish();
#endif

#ifdef OPENME
  openme_callback("PROGRAM_END", NULL);
#endif

  return 0;
}

