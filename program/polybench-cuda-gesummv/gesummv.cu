#include "hip/hip_runtime.h"
/**
 * gesummv.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 *
 * Updated by Grigori Fursin (http://cTuning.org/lab/people/gfursin)
 * to work with Collective Mind, OpenME plugin interface and 
 * Collective Knowledge Frameworks for automatic, machine-learning based
 * and collective tuning and data mining: http://cTuning.org
 *
 */

#ifndef WINDOWS
 #include <unistd.h>
#endif

#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>

#include <hip/hip_runtime.h>

#include "polybench.h"

#ifdef OPENME
#include <openme.h>
#endif
#ifdef XOPENME
#include <xopenme.h>
#endif

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size */
#ifndef NJ
#define NJ 256 //4096
#endif

/* Thread block dimensions */
#ifndef DIM_THREAD_BLOCK_X
#define DIM_THREAD_BLOCK_X 256
#endif
#ifndef DIM_THREAD_BLOCK_Y
#define DIM_THREAD_BLOCK_Y 1
#endif

/* Declared constant values for ALPHA and BETA (same as values in PolyBench 2.0) */
#define ALPHA 43532.0f
#define BETA 12313.0f

/* Can switch DATA_TYPE between float and double */
# ifndef DATA_TYPE
#  define DATA_TYPE float
# endif

void gesummv(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *x, DATA_TYPE *y, DATA_TYPE *tmp)
{
	int i, j;
	
	for (i = 0; i < NJ; i++)
	{
		tmp[i] = 0;
		y[i] = 0;
		for (j = 0; j < NJ; j++)
		{
			tmp[i] = A[i*NJ + j] * x[j] + tmp[i];
			y[i] = B[i*NJ + j] * x[j] + y[i];
		}
		
		y[i] = ALPHA * tmp[i] + BETA * y[i];
	}
}


void init(DATA_TYPE* A, DATA_TYPE* x)
{
  	int i, j;

 	for (i = 0; i < NJ; i++)
    {
    	x[i] = ((DATA_TYPE) i) / NJ;
      	
		for (j = 0; j < NJ; j++) 
		{
			A[i*NJ + j] = ((DATA_TYPE) i*j) / NJ;
		}
    }
}


void compareResults(DATA_TYPE* y, DATA_TYPE* y_outputFromGpu)
{
	int i, fail;
	fail = 0;
	
	for (i=0; i<(NJ); i++) 
	{
		if (percentDiff(y[i], y_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD) 
		{
			fail++;
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
  /* Grigori Fursin added support for CK widgets */
  int gpgpu_device_id=GPU_DEVICE;

  int devID = 0;
  hipError_t error;
  hipDeviceProp_t deviceProp;
  error = hipGetDevice(&devID);

  if (getenv("CK_COMPUTE_DEVICE_ID")!=NULL) gpgpu_device_id=atol(getenv("CK_COMPUTE_DEVICE_ID"));

  hipGetDeviceProperties(&deviceProp, gpgpu_device_id);

  if (deviceProp.computeMode == hipComputeModeProhibited)
  {
    printf("Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
    exit(EXIT_SUCCESS);
  }

  if (error != hipSuccess)
    printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
  else
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);

  hipSetDevice( gpgpu_device_id );
}


__global__ void gesummv_kernel(DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *x, DATA_TYPE *y, DATA_TYPE *tmp)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < NJ)
	{
		int j;
		y[i]=0;
		for(j = 0; j < NJ; j++)
		{	
			tmp[i] += a[i * NJ + j] * x[j];
			y[i] += b[i * NJ + j] * x[j];
		}
		y[i] = ALPHA * tmp[i] + BETA * y[i];
	}
}

void gesummvCuda(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* x, DATA_TYPE* y, DATA_TYPE* tmp, DATA_TYPE* y_outputFromGpu)
{
        hipError_t error;
	double t_start, t_end;		

	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *x_gpu;
	DATA_TYPE *y_gpu;
	DATA_TYPE *tmp_gpu;

	error=hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NJ * NJ);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NJ * NJ);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&x_gpu, sizeof(DATA_TYPE) * NJ);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&y_gpu, sizeof(DATA_TYPE) * NJ);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&tmp_gpu, sizeof(DATA_TYPE) * NJ);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }
	
	error=hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NJ * NJ, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NJ * NJ, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(x_gpu, x, sizeof(DATA_TYPE) * NJ, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(y_gpu, y, sizeof(DATA_TYPE) * NJ, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(tmp_gpu, tmp, sizeof(DATA_TYPE) * NJ, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((unsigned int)ceil( ((float)NJ) / ((float)block.x) ), 1);


//	t_start = rtclock();
	gesummv_kernel<<< grid, block>>>(A_gpu,B_gpu,x_gpu, y_gpu, tmp_gpu);
	hipDeviceSynchronize();
//	t_end = rtclock();
	error=hipMemcpy(y_outputFromGpu, y_gpu, sizeof(DATA_TYPE) * NJ, hipMemcpyDeviceToHost);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

//	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
}


int main(int argc, char *argv[])
{
  /* Prepare ctuning vars */
  long ct_repeat=0;
  long ct_repeat_max=1;

  double t_start, t_end;

  DATA_TYPE* A;
  DATA_TYPE* B;  
  DATA_TYPE* x;  
  DATA_TYPE* y;
  DATA_TYPE* y_outputFromGpu;
  DATA_TYPE* tmp;

#ifdef XOPENME
  xopenme_init(2,0);
#endif

#ifdef OPENME
  openme_init(NULL,NULL,NULL,0);
  openme_callback("PROGRAM_START", NULL);
#endif

  /* Run kernel. */
  if (getenv("CT_REPEAT_MAIN")!=NULL) ct_repeat_max=atol(getenv("CT_REPEAT_MAIN"));

  A = (DATA_TYPE*)malloc(NJ*NJ*sizeof(DATA_TYPE));
  B = (DATA_TYPE*)malloc(NJ*NJ*sizeof(DATA_TYPE));
  x = (DATA_TYPE*)malloc(NJ*sizeof(DATA_TYPE)); 
  y = (DATA_TYPE*)malloc(NJ*sizeof(DATA_TYPE));
  y_outputFromGpu = (DATA_TYPE*)malloc(NJ*sizeof(DATA_TYPE));
  tmp = (DATA_TYPE*)malloc(NJ*sizeof(DATA_TYPE));

  srand(1);
  init(A, x);
  GPU_argv_init();

#ifdef OPENME
  openme_callback("ACC_KERNEL_START", NULL);
#endif
#ifdef XOPENME
  xopenme_clock_start(0);
#endif
  for (ct_repeat=0; ct_repeat<ct_repeat_max; ct_repeat++)
  {
    gesummvCuda(A, B, x, y, tmp, y_outputFromGpu);
  }
#ifdef XOPENME
  xopenme_clock_end(0);
#endif
#ifdef OPENME
  openme_callback("ACC_KERNEL_END", NULL);
#endif

/*
  srand(1);
  init(A, x);

#ifdef OPENME
  openme_callback("KERNEL_START", NULL);
#endif
#ifdef XOPENME
  xopenme_clock_start(1);
#endif
  for (ct_repeat=0; ct_repeat<ct_repeat_max; ct_repeat++)
  {
    gesummv(A, B, x, y, tmp);
  }
#ifdef XOPENME
  xopenme_clock_end(1);
#endif
#ifdef OPENME
  openme_callback("KERNEL_END", NULL);
#endif
*/

  compareResults(y, y_outputFromGpu);

  free(A);
  free(B);
  free(x);
  free(y);
  free(y_outputFromGpu);
  free(tmp);

#ifdef XOPENME
  xopenme_dump_state();
  xopenme_finish();
#endif

#ifdef OPENME
  openme_callback("PROGRAM_END", NULL);
#endif

  return 0;
}

