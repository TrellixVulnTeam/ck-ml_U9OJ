#include "hip/hip_runtime.h"
/**
 * 2mm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 *
 * Updated by Grigori Fursin (http://cTuning.org/lab/people/gfursin)
 * to work with Collective Mind, OpenME plugin interface and 
 * Collective Knowledge Frameworks for automatic, machine-learning based
 * and collective tuning and data mining: http://cTuning.org
 *
 */

#ifndef WINDOWS
 #include <unistd.h>
#endif

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include "polybench.h"

#ifdef OPENME
#include <openme.h>
#endif
#ifdef XOPENME
#include <xopenme.h>
#endif

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size. */
#ifndef NI
# define NI 256 //2048
#endif
#ifndef NJ
# define NJ 256 //2048
#endif
#ifndef NK
# define NK 256 //2048
#endif
#ifndef NL
# define NL 256 //2048
#endif

/* Thread block dimensions */
#ifndef DIM_TRHEAD_BLOCK_X
#define DIM_THREAD_BLOCK_X 8 //32
#endif
#ifndef DIM_THREAD_BLOCK_Y
#define DIM_THREAD_BLOCK_Y 8
#endif

/* Can switch DATA_TYPE between float and double */
# ifndef DATA_TYPE
#  define DATA_TYPE float
# endif

void init_array(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D)
{
	int i, j;

	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NK; j++)
		{
			A[i*NI + j] = ((DATA_TYPE) i*j) / NI;
		}
	}

	for (i = 0; i < NK; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			B[i*NK + j] = ((DATA_TYPE) i*(j+1)) / NJ;
		}
	}

	for (i = 0; i < NL; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			C[i*NL + j] = ((DATA_TYPE) i*(j+3)) / NL;
		}
	}

	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NL; j++)
		{
			D[i*NL + j] = ((DATA_TYPE) i*(j+2)) / NK;	
		}
	}
}


void compareResults(DATA_TYPE *E, DATA_TYPE *E_outputFromGpu)
{
	int i,j,fail;
	fail = 0;

	for (i=0; i < NL; i++)
	{
		for (j=0; j < NI; j++)
		{
			if (percentDiff(E[i*NI + j], E_outputFromGpu[i*NI + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{
				fail++;
			}
		}
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
  /* Grigori Fursin added support for CK widgets */
  int gpgpu_device_id=GPU_DEVICE;

  int devID = 0;
  hipError_t error;
  hipDeviceProp_t deviceProp;
  error = hipGetDevice(&devID);

  if (getenv("CK_COMPUTE_DEVICE_ID")!=NULL) gpgpu_device_id=atol(getenv("CK_COMPUTE_DEVICE_ID"));

  hipGetDeviceProperties(&deviceProp, gpgpu_device_id);

  if (deviceProp.computeMode == hipComputeModeProhibited)
  {
    printf("Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
    exit(EXIT_SUCCESS);
  }

  if (error != hipSuccess)
    printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
  else
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);

  hipSetDevice( gpgpu_device_id );
}


__global__ void mm2_kernel1(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NJ))
	{ 
		int k;
		for (k = 0; k < NK; k++)
		{
			C[i * NJ + j] += A[i * NK + k] * B[k * NJ + j];
		}
	}
}


__global__ void mm2_kernel2(DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *E)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NL))
	{ 
		int k;
		for (k = 0; k < NJ; k++)
		{
			E[i * NL + j] += C[i * NJ + k] * D[k * NL + j];
		}
	}
}


void mm2_cpu(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, DATA_TYPE* E)
{
	int i, j, k;
	
  	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			C[i*NJ + j] = 0.0;
			for (k = 0; k < NK; ++k)
			{
				C[i*NJ + j] += A[i*NK + k] * B[k*NJ + j];
			}
		}
	}
	
	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NL; j++)
		{
			E[i*NL + j] = 0.0;
			for (k = 0; k < NJ; ++k)
			{
				E[i*NL + j] += C[i*NJ + k] * D[k*NL + j];
			}
		}
	}
}


void mm2Cuda(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, DATA_TYPE* E, DATA_TYPE* E_outputFromGpu)
{
        hipError_t error;
	double t_start, t_end;

	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;
	DATA_TYPE *D_gpu;
	DATA_TYPE *E_gpu;

	error=hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NI * NJ);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&D_gpu, sizeof(DATA_TYPE) * NJ * NL);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&E_gpu, sizeof(DATA_TYPE) * NI * NL);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }
	
	error=hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(D_gpu, D, sizeof(DATA_TYPE) * NJ * NL, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(E_gpu, E, sizeof(DATA_TYPE) * NI * NL, hipMemcpyHostToDevice);	
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }
		
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid1((size_t)ceil( ((float)NJ) / ((float)block.x) ), (size_t)ceil( ((float)NI) / ((float)block.y)) );
	dim3 grid2((size_t)ceil( ((float)NL) / ((float)block.x) ), (size_t)ceil( ((float)NI) / ((float)block.y)) );
//	t_start = rtclock();
	mm2_kernel1<<<grid1,block>>>(A_gpu, B_gpu, C_gpu);
	hipDeviceSynchronize();
	mm2_kernel2<<<grid2,block>>>(C_gpu, D_gpu, E_gpu);
	hipDeviceSynchronize();
//	t_end = rtclock();
//	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

	error=hipMemcpy(E_outputFromGpu, E_gpu, sizeof(DATA_TYPE) * NI * NL, hipMemcpyDeviceToHost);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
	hipFree(D_gpu);
	hipFree(E_gpu);
}


int main(int argc, char** argv)
{
  /* Prepare ctuning vars */
  long ct_repeat=0;
  long ct_repeat_max=1;

  DATA_TYPE* C;
  DATA_TYPE* A;
  DATA_TYPE* B;
  DATA_TYPE* D;
  DATA_TYPE* E;
  DATA_TYPE* E_outputFromGpu;

#ifdef XOPENME
  xopenme_init(2,0);
#endif

#ifdef OPENME
  openme_init(NULL,NULL,NULL,0);
  openme_callback("PROGRAM_START", NULL);
#endif

  /* Run kernel. */
  if (getenv("CT_REPEAT_MAIN")!=NULL) ct_repeat_max=atol(getenv("CT_REPEAT_MAIN"));

  C = (DATA_TYPE*)malloc(NI*NJ*sizeof(DATA_TYPE));
  A = (DATA_TYPE*)malloc(NI*NK*sizeof(DATA_TYPE));
  B = (DATA_TYPE*)malloc(NK*NJ*sizeof(DATA_TYPE));
  D = (DATA_TYPE*)malloc(NJ*NL*sizeof(DATA_TYPE));
  E = (DATA_TYPE*)malloc(NI*NL*sizeof(DATA_TYPE));
  E_outputFromGpu = (DATA_TYPE*)malloc(NI*NL*sizeof(DATA_TYPE));

  srand(1);
  init_array(A, B, C, D);
  GPU_argv_init();

#ifdef OPENME
  openme_callback("ACC_KERNEL_START", NULL);
#endif
#ifdef XOPENME
  xopenme_clock_start(0);
#endif
  for (ct_repeat=0; ct_repeat<ct_repeat_max; ct_repeat++)
  {
    mm2Cuda(A, B, C, D, E, E_outputFromGpu);
  }
#ifdef XOPENME
  xopenme_clock_end(0);
#endif
#ifdef OPENME
  openme_callback("ACC_KERNEL_END", NULL);
#endif

/*
  srand(1);
  init_array(A, B, C, D);

#ifdef OPENME
  openme_callback("KERNEL_START", NULL);
#endif
#ifdef XOPENME
  xopenme_clock_start(1);
#endif
  for (ct_repeat=0; ct_repeat<ct_repeat_max; ct_repeat++)
  {
    mm2_cpu(A, B, C, D, E);
  }
#ifdef XOPENME
  xopenme_clock_end(1);
#endif
#ifdef OPENME
  openme_callback("KERNEL_END", NULL);
#endif
*/

  compareResults(E, E_outputFromGpu);

  free(C);
  free(A);
  free(B);
  free(D);
  free(E);
  free(E_outputFromGpu);

#ifdef XOPENME
  xopenme_dump_state();
  xopenme_finish();
#endif

#ifdef OPENME
  openme_callback("PROGRAM_END", NULL);
#endif

  return 0;
}

